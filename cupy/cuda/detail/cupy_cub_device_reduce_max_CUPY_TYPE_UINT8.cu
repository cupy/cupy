#include "../cupy_cub.inl"


namespace cupy {

void cub_device_reduce_max_CUPY_TYPE_UINT8(void* workspace,
                                      size_t& workspace_size,
                                      void* x,
                                      void* y,
                                      int num_items,
                                      hipStream_t stream) {
#if ( CUPY_TYPE_UINT8 != CUPY_TYPE_FLOAT16 )                        \
    || (( CUPY_TYPE_UINT8 == CUPY_TYPE_FLOAT16 )                    \
        && ((__CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__))  \
            || (defined(__HIPCC__) || defined(CUPY_USE_HIP))))

    _cub_reduce_max op;
    return dtype_forwarder< unsigned char >(op,
                                         workspace,
                                         workspace_size,
                                         x,
                                         y,
                                         num_items,
                                         stream);
#endif
}

}  // namespace cupy
