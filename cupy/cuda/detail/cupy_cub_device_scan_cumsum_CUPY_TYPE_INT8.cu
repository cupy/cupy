#include "../cupy_cub.inl"


namespace cupy {

void cub_device_scan_cumsum_CUPY_TYPE_INT8(void* workspace,
                                       size_t& workspace_size,
                                       void* x,
                                       void* y,
                                       int num_items,
                                       hipStream_t stream) {
#if ( CUPY_TYPE_INT8 != CUPY_TYPE_FLOAT16 )                        \
    || (( CUPY_TYPE_INT8 == CUPY_TYPE_FLOAT16 )                    \
        && ((__CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__))  \
            || (defined(__HIPCC__) || defined(CUPY_USE_HIP))))

    _cub_inclusive_sum op;
    return dtype_forwarder< char >(op,
                                         workspace,
                                         workspace_size,
                                         x,
                                         y,
                                         num_items,
                                         stream);
#endif
}

}  // namespace cupy
